#include "hip/hip_runtime.h"
#include "GPU_ma_rock_sample.h"
#include "base_ma_rock_sample.h"
#include "ma_rock_sample.h"
#include <bitset>

#include <despot/solver/Hyp_despot.h>

#include <despot/GPUinterface/GPUupper_bound.h>

using namespace std;

namespace despot {

/* =============================================================================
 * Dvc_MultiAgentRockSample class
 * =============================================================================*/
extern __shared__ int localParticles[];

__global__ void step_global( Dvc_State* vnode_particles,float* rand,
		float * reward, OBS_TYPE* obs, float* ub, Dvc_ValuedAction* lb,
		bool* term, int num_particles, int parent_action, Dvc_State* state)
{
	int action=blockIdx.x;
	int PID = (blockIdx.y * blockDim.x + threadIdx.x) ;
	if(PID<num_particles){
		int global_list_pos = action * num_particles + PID;
		float rand_num=rand[global_list_pos];

		if (threadIdx.y == 0) {
			DvcModelCopyToShared_(
				(Dvc_State*) ((int*) localParticles + 8 * threadIdx.x),
				vnode_particles, PID % num_particles, false);
		}
		Dvc_State* current_particle = (Dvc_State*) ((int*) localParticles + 8 * threadIdx.x);
		__syncthreads();


		term[global_list_pos]=DvcModelStep_(*current_particle, rand_num, parent_action, reward[global_list_pos], obs[global_list_pos]);

		if (blockIdx.y * blockDim.x + threadIdx.x < num_particles) {
			/*Record stepped particles from parent as particles in this node*/
			if (threadIdx.y == 0 && action==0) {
				Dvc_State* temp = DvcModelGet_(vnode_particles, PID % num_particles);
				DvcModelCopyNoAlloc_(temp, current_particle,0, false);
			}
		}

		term[global_list_pos]=DvcModelStep_(*current_particle, rand_num, action, reward[global_list_pos], obs[global_list_pos]);

		Dvc_History history;
		Dvc_RandomStreams streams;
		ub[global_list_pos]=DvcUpperBoundValue_(current_particle, 0, history);
		lb[global_list_pos]=DvcLowerBoundValue_(current_particle,streams,history, 0) ;

		Dvc_State* temp = DvcModelGet_(state, global_list_pos);
		DvcModelCopyNoAlloc_(temp, current_particle, 0, false);
	}
}


__global__ void step_global_1( Dvc_State* vnode_particles,float* rand,
		float * reward, OBS_TYPE* obs, float* ub, Dvc_ValuedAction* lb,
		bool* term, int num_particles, int parent_action, Dvc_State* state)
{
	int action=blockIdx.x;
	int PID = (blockIdx.y * blockDim.x + threadIdx.x) ;
	if(PID<num_particles){
		int global_list_pos = action * num_particles + PID;
		float rand_num=rand[global_list_pos];

		if (threadIdx.y == 0) {
			DvcModelCopyToShared_(
				(Dvc_State*) ((int*) localParticles + 8 * threadIdx.x),
				vnode_particles, PID % num_particles, false);
		}
		Dvc_State* current_particle = (Dvc_State*) ((int*) localParticles + 8 * threadIdx.x);
		__syncthreads();

		term[global_list_pos]=DvcModelStep_(*current_particle, rand_num, parent_action, reward[global_list_pos], obs[global_list_pos]);


		Dvc_State* temp = DvcModelGet_(state, global_list_pos);
		DvcModelCopyNoAlloc_(temp, current_particle, 0, false);
	}
}
__global__ void step_global_2( Dvc_State* vnode_particles,float* rand,
		float * reward, OBS_TYPE* obs, float* ub, Dvc_ValuedAction* lb,
		bool* term, int num_particles, int parent_action, Dvc_State* state)
{
	int action=blockIdx.x;
	int PID = (blockIdx.y * blockDim.x + threadIdx.x) ;
	if(PID<num_particles){
		int global_list_pos = action * num_particles + PID;
		float rand_num=rand[global_list_pos];

		if (threadIdx.y == 0) {
			DvcModelCopyToShared_(
				(Dvc_State*) ((int*) localParticles + 8 * threadIdx.x),
				state, global_list_pos, false);
		}
		Dvc_State* current_particle = (Dvc_State*) ((int*) localParticles + 8 * threadIdx.x);


		if (blockIdx.y * blockDim.x + threadIdx.x < num_particles) {
			/*Record stepped particles from parent as particles in this node*/
			if (threadIdx.y == 0 && action==0) {
				Dvc_State* temp = DvcModelGet_(vnode_particles, PID % num_particles);
				DvcModelCopyNoAlloc_(temp, current_particle,0, false);
			}
		}
		__syncthreads();


		term[global_list_pos]=DvcModelStep_(*current_particle, rand_num, action, reward[global_list_pos], obs[global_list_pos]);

		Dvc_History history;
		Dvc_RandomStreams streams;
		ub[global_list_pos]=DvcUpperBoundValue_(current_particle, 0, history);
		lb[global_list_pos]=DvcLowerBoundValue_(current_particle,streams,history, 0) ;

		Dvc_State* temp = DvcModelGet_(state, global_list_pos);
		DvcModelCopyNoAlloc_(temp, current_particle, 0, false);
	}
}

DEVICE bool Dvc_MultiAgentRockSample::Dvc_Step(Dvc_State& state, float rand_num, int action, float& reward,
			OBS_TYPE& obs)
{
	reward=0;
	obs=0;
	bool terminal=true;
	Dvc_MARockSampleState& rockstate = static_cast<Dvc_MARockSampleState&>(state);

	__syncthreads();

	for(int rid=0;rid<num_agents_;rid++)
	{
		SetRobObs(obs, E_NONE, rid);

		if(GetRobPosIndex(&rockstate, rid)!=ROB_TERMINAL_ID){

			int rob_act=GetRobAction(action, rid);
			//rob_act=Dvc_Compass::EAST;//debugging
			if (rob_act < E_SAMPLE) { // Move
				switch (rob_act) {
				case Dvc_Compass::EAST:
					if (GetX(&rockstate, rid) + 1 < ma_map_size_) {
						IncX(&rockstate, rid);
					} else {
						reward+= +10;
						SetRobPosIndex(rockstate.joint_pos, rid, ROB_TERMINAL_ID);
					}
					break;

				case Dvc_Compass::NORTH:
					if (GetY(&rockstate, rid) + 1 < ma_map_size_)
						IncY(&rockstate, rid);
					else{
						reward += -100;
					}
					break;

				case Dvc_Compass::SOUTH:
					if (GetY(&rockstate, rid) - 1 >= 0)
						DecY(&rockstate, rid);
					else
						reward += -100;
					break;

				case Dvc_Compass::WEST:
					if (GetX(&rockstate, rid) - 1 >= 0)
						DecX(&rockstate, rid);
					else
						reward += -100;
					break;
				}
			}
			if (rob_act == E_SAMPLE) { // Sample
				int rock = ma_grid_[GetRobPosIndex(&rockstate, rid)];
				if (rock >= 0) {
					if (GetRock(&rockstate, rock))
						reward += +10;
					else
						reward += -10;
					SampleRock(&rockstate, rock);
				} else {
					reward += -100;
				}
			}

			if (rob_act > E_SAMPLE) { // Sense
				int rock = rob_act - E_SAMPLE - 1;
				float distance = DvcCoord::EuclideanDistance(GetRobPos(&rockstate, rid),
					ma_rock_pos_[rock]);
				float efficiency = (1 + powf(2, -distance / ma_half_efficiency_distance_))
					* 0.5;

				if (rand_num < efficiency)
					SetRobObs(obs, GetRock(&rockstate, rock) & E_GOOD, rid);
				else
					SetRobObs(obs, !(GetRock(&rockstate, rock) & E_GOOD), rid);
			}


			if (GetRobPosIndex(&rockstate, rid)!=ROB_TERMINAL_ID) {
				terminal=false;
			}
		}
	}

	if(GPUDoPrint/* && action==blockIdx.x*/)
			printf("(GPU_step) action %d scenario %d state_id %d joint_pos %d blockid.y %d threadid.x %d rand %f\n",
				action, rockstate.scenario_id, rockstate.state_id, rockstate.joint_pos, blockIdx.y, threadIdx.x, rand_num);

	return terminal;
}

DEVICE int Dvc_MultiAgentRockSample::NumActions()
{
	return pow((float)(ma_num_rocks_ + 5), num_agents_);
}


DEVICE int Dvc_MultiAgentRockSample::Dvc_NumObservations()
{
	return /*3*/num_agents_*MAX_OBS_BIT;
}
DEVICE Dvc_State* Dvc_MultiAgentRockSample::Dvc_Get(Dvc_State* particles, int pos)
{
	Dvc_MARockSampleState* particle_i= static_cast<Dvc_MARockSampleState*>(particles)+pos;

	return particle_i;
}
DEVICE void Dvc_MultiAgentRockSample::Dvc_Copy_NoAlloc(Dvc_State* des, const Dvc_State* src, int pos, bool offset_des)
{
	/*Pass member values, assign member pointers to existing state pointer*/
	const Dvc_MARockSampleState* src_i= static_cast<const Dvc_MARockSampleState*>(src)+pos;
	if(!offset_des) pos=0;
	Dvc_MARockSampleState* des_i= static_cast<Dvc_MARockSampleState*>(des)+pos;

	des_i->weight = src_i->weight;
	des_i->scenario_id = src_i->scenario_id;
	des_i->state_id = src_i->state_id;
	des_i->joint_pos = src_i->joint_pos;

	//des_i->allocated_=true;
}

} // namespace despot
